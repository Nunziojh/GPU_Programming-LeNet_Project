#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define h_m 32
#define w_m 32
#define leakyReluSlope 0.1

__global__ void sigmoid(float *in){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;

    if(idx < w_m && idy < h_m){

        in[idy * w_m + idx] = 1 / (1 + exp(in[idy * w_m + idx]));
    }
}

__global__ void tanh(float *in){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;

    if(idx < w_m && idy < h_m){

        in[idy * w_m + idx] = 1 / (1 + exp(in[idy * w_m + idx]));
    }
}

__global__ void relu(float *in){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;

    if(idx < w_m && idy < h_m){

        in[idy * w_m + idx] = max(0.0, in[idy * w_m + idx]);
    }
}

__global__ void leakyRelu(float *in){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;

    if(idx < w_m && idy < h_m){

        max(0.0, in[idy * w_m + idx]) + (-leakyReluSlope) * min(0.0, in[idy * w_m + idx]);

        in[idy * w_m + idx] = max(, in[idy * w_m + idx]);
    }
}

int main(int argc, char **argv){

    float *host = (float *)malloc(sizeof(float) * h_m * w_m);
    for(int i = 0; i < h_m * w_m; i++) host[i] = i;

    float *dev;
    hipMalloc((void **)&dev, h_m * w_m * sizeof(float));
    hipMemcpy(dev, host, sizeof(float) * h_m * w_m, hipMemcpyHostToDevice);
    
    dim3 block = {32, 32};
    dim3 grid = {w_m / block.x + 1, h_m / block.y + 1};

    sigmoid<<<grid, block>>>(dev);

    hipMemcpy(host, dev, sizeof(float) * h_m * w_m, hipMemcpyDeviceToHost);

    for(int i = 0; i < h_m; i++){
        for(int j = 0; j < w_m; j++){
            printf("%2.2f ", host[i * w_m + j]);
        }
        printf("\n");
    }

    free(host);
    hipFree(dev);

    return 0;

}