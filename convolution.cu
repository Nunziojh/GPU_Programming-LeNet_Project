#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>


#define h_k 3
#define w_k 3
#define h_m 5
#define w_m 5
#define stride 1
#define padding 0

__constant__ int kernel[h_k * w_k];

__global__ void kernel_function(int *in, int *out, int new_h, int new_w){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;

    if(idx < new_w && idy < new_h){

        int i, j;
        int r = h_k / 2;
        int c = w_k / 2;

        int tmp = 0;
        int val;

        int new_idx = idx * stride + c - padding;
        int new_idy = idy * stride + r - padding;

        for(i = -r; i <= r; i++){
            for(j = -c; j <= c; j++){
                val = ((new_idy + i) < 0 || (new_idy + i) >= h_m || (new_idx + j) < 0 || (new_idx + j) >= w_m) ? 0 : in[(new_idy + i) * w_m + new_idx + j];
                tmp += kernel[(r-i) * w_k + (c-j)] * val;
            }
        }
        out[idy * new_w + idx] = tmp;
    }
}



int main(int argc, char **argv){

    srand(time(NULL));

    int host_matrix[] = {1, 5, 2, 3, 6,
                        7, 10, 2, 8, 4,
                        10, 6, 5, 4, 3,
                        2, 2, 2, 1, 1,
                        1, 6, 7, 8, 9};
    int host_kernel[] = {1, 1, 1,
                        0, 0, 0,
                        -1, -1, -1};
    //for(int i = 0; i< w_k * h_k; i++) host_kernel[i] = 1;
    //int *host_input = (int *)malloc(sizeof(int) * h_m * w_m);
    
    int new_h = (h_m + 2 * padding - h_k) / stride + 1;
    int new_w = (w_m + 2 * padding - w_k) / stride + 1;
    int *host_res = (int *)malloc(sizeof(int) * new_h * new_w);

    int *dev_input, *dev_res;
    hipMalloc((void **)&dev_input, h_m * w_m * sizeof(int));
    hipMalloc((void **)&dev_res, new_h * new_w * sizeof(int));
    hipMemcpy(dev_input, host_matrix, sizeof(int) * h_m * w_m, hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(kernel), &host_kernel, sizeof(int) * w_k * h_k);
    
    dim3 block = {32, 32};
    dim3 grid = {new_w / block.x + 1, new_h / block.y + 1};

    kernel_function<<<grid, block>>>(dev_input, dev_res, new_h, new_w);

    hipMemcpy(host_res, dev_res, sizeof(int) * new_h * new_w, hipMemcpyDeviceToHost);

    for(int i = 0; i < new_h; i++){
        for(int j = 0; j < new_w; j++){
            printf("%d ", host_res[i * new_w + j]);
        }
        printf("\n");
    }

    //free(host_matrix);
    free(host_res);
    hipFree(dev_input);
    hipFree(dev_res);

    return 0;

}