#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>


#define h_k 3
#define w_k 3
#define h_m 5
#define w_m 5
#define stride 1
#define padding 1

__constant__ int kernel[h_k * w_k];

__global__ void kernel_function(int *in, int *out, int new_h, int new_w){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;

    if(idx < new_w && idy < new_h){

        int i, j;
        int r = h_k / 2;
        int c = w_k / 2;

        int tmp = 0;
        int val;

        int new_idx = idx * stride - c + padding;
        int new_idy = idy * stride - r + padding;

        for(i = 0; i < h_k; i++){
            for(j = 0; j < w_k; j++){
                val = ((new_idy + i) < 0 || (new_idy + i) >= h_m || (new_idx + j) < 0 || (new_idx + j) >= w_m) ? 0 : in[(new_idy + i) * w_m + new_idx + j];
                tmp += kernel[i * w_k + j] * val;
            }
        }
        printf("(%02d, %02d)\t%d\n", new_idy, new_idx, tmp);
        out[idy * new_w + idx] = tmp;
    }
}

int main(int argc, char **argv){

    srand(time(NULL));

    int host_kernel[w_k * h_k] = {1, 1, 1, 1, 1, 1, 1, 1, 1};

    int *host_input = (int *)malloc(sizeof(int) * h_m * w_m);
    for(int i = 0; i < h_m * w_m; i++) host_input[i] = i;
    int new_h = (h_m + 2 * padding - h_k) / stride + 1;
    int new_w = (w_m + 2 * padding - w_k) / stride + 1;
    int *host_res = (int *)malloc(sizeof(int) * new_h * new_w);

    int *dev_input, *dev_res;
    hipMalloc((void **)&dev_input, h_m * w_m * sizeof(int));
    hipMalloc((void **)&dev_res, new_h * new_w * sizeof(int));
    hipMemcpy(dev_input, host_input, sizeof(int) * h_m * w_m, hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(kernel), &host_kernel, sizeof(int) * w_k * h_k);
    
    dim3 block = {32, 32};
    dim3 grid = {new_w / block.x + 1, new_h / block.y + 1};

    kernel_function<<<grid, block>>>(dev_input, dev_res, new_h, new_w);

    hipMemcpy(host_res, dev_res, sizeof(int) * new_h * new_w, hipMemcpyDeviceToHost);

    for(int i = 0; i < new_h; i++){
        for(int j = 0; j < new_w; j++){
            printf("%d ", host_res[i * new_w + j]);
        }
        printf("\n");
    }

    free(host_input);
    free(host_res);
    hipFree(dev_input);
    hipFree(dev_res);

    return 0;

}