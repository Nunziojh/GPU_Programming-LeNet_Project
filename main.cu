#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>


#define DIM 100

__global__ void kernel_function(int *in_vec){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;
    int idz = blockDim.z * blockIdx.z + threadIdx.z;

    in_vec[idx * 5 + idy + idz * 25] = in_vec[idx * 5 + idy + idz * 25] * 2;
}

int main(int argc, char **argv){

    dim3 grid = {2};
    dim3 block = {5, 5, 2};

    int *memDev;
    int *input_vect = (int *) malloc(sizeof(int) * DIM);

    for(int i = 0; i < DIM; i++) input_vect[i] = i;

    
    hipMalloc((void **)&memDev, DIM * sizeof(int));

    hipMemcpy(memDev, input_vect, sizeof(int) * DIM, hipMemcpyHostToDevice);

    kernel_function<<<grid, block>>>(memDev);

    hipMemcpy(input_vect, memDev, sizeof(int) * DIM, hipMemcpyDeviceToHost);

    for(int i = 0; i < DIM; i++) printf("%d\n", input_vect[i]);

    free(input_vect);
    hipFree(memDev);

    return 0;

}